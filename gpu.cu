#include "hip/hip_runtime.h"
#include "gpu.cuh"

#include <iostream>

#define MAX_MASS 150
#define MAX_COMPRESS 0.5
#define MIN_MASS 1
#define FLOW_STRENGHT 1

__device__ void SetPixel(uint8_t* surfaceData, int index, uint8_t r, uint8_t g, uint8_t b) {
	surfaceData[index * 4] = b;
	surfaceData[index * 4 + 1] = g;
	surfaceData[index * 4 + 2] = r;
}

__device__ int clamp(int a, int min, int max) {
	return a > max ? max : a < min ? min : a;
}

__device__ uint8_t GetType(uint8_t* simData, int index) {
	return simData[index * 2];
}

__device__ uint8_t GetMass(uint8_t* simData, int index) {
	return simData[index * 2 + 1];
}

__device__ void SetType(uint8_t* simData, int index, uint8_t type) {
	simData[index * 2] = type;
}

__device__ void SetMass(uint8_t* simData, int index, uint8_t mass) {
	simData[index * 2 + 1] = mass;
}

__device__ void AddMass(uint8_t* simData, int index, uint8_t mass) {
	simData[index * 2 + 1] += mass;
}

__global__ void DrawGPU(uint8_t* surfaceData, uint8_t* simData) {
	int currentIndex = threadIdx.x + (blockIdx.x * blockDim.x);

	switch (simData[currentIndex * 2]) {
	case 1: {
		SetPixel(surfaceData, currentIndex, 0, 0, GetMass(simData, currentIndex));
		break;
	}
	case 2: {
		SetPixel(surfaceData, currentIndex, 168, 96, 50);
		break;
	}
	default: {
		SetPixel(surfaceData, currentIndex, 0, 0, 0);
		break;
	}
	}
}

__global__ void ComputeGPU(uint8_t* curSimData, uint8_t* nextSimData, int* size) {
	int w = size[0];
	int h = size[1];

	int currentIndex = threadIdx.x + (blockIdx.x * blockDim.x);
	int belowIndex = currentIndex + w;
	int aboveIndex = currentIndex - w;
	int rightIndex = currentIndex + 1;
	int leftIndex = currentIndex - 1;

	int rightBelowIndex = currentIndex + 1 + w;
	int rightAboveIndex = currentIndex + 1 - w;
	int leftBelowIndex = currentIndex - 1 + w;
	int leftAboveIndex = currentIndex - 1 - w;

	if (GetType(curSimData, currentIndex) > 1) {
		SetType(nextSimData, currentIndex, GetType(curSimData, currentIndex));
		SetMass(nextSimData, currentIndex, GetMass(curSimData, currentIndex));
		return;
	}

	uint8_t currentMass = GetMass(curSimData, currentIndex);

	SetMass(nextSimData, currentIndex, currentMass);

	if (GetType(curSimData, aboveIndex) < 2) {
		uint8_t aboveMass = GetMass(curSimData, aboveIndex);
		uint8_t currentCapacity = clamp((MAX_MASS + (int)aboveMass * MAX_COMPRESS) - (int)currentMass, 0, 255);
		int flow = clamp(currentCapacity, -currentMass, currentMass + aboveMass <= 255 ? aboveMass : 255 - currentMass);
		flow *= FLOW_STRENGHT;
		AddMass(nextSimData, currentIndex, flow);
	}

	int belowFlow = 0;

	if (GetType(curSimData, belowIndex) < 2) {
		uint8_t belowMass = GetMass(curSimData, belowIndex);
		uint8_t belowCapacity = clamp((MAX_MASS + (int)currentMass * MAX_COMPRESS) - (int)belowMass, 0, 255);
		belowFlow = clamp(belowCapacity, -belowMass, belowMass + currentMass <= 255 ? currentMass : 255 - belowMass);
		belowFlow *= FLOW_STRENGHT;
		AddMass(nextSimData, currentIndex, -belowFlow);
	}

	uint8_t rightMass = GetMass(curSimData, rightIndex);
	uint8_t leftMass = GetMass(curSimData, leftIndex);

	uint8_t newCurrentMass = GetMass(nextSimData, currentIndex);
	uint8_t newRightMass = rightMass;
	uint8_t newLeftMass = leftMass;
	//right Update

	if (GetType(curSimData, rightAboveIndex) < 2) {
		uint8_t rightAboveMass = GetMass(curSimData, rightAboveIndex);
		uint8_t rightCurrentCapacity = clamp((MAX_MASS + (int)rightAboveMass * MAX_COMPRESS) - (int)rightMass, 0, 255);
		int flow = clamp(rightCurrentCapacity, -rightMass, rightMass + rightAboveMass <= 255 ? rightAboveMass : 255 - rightMass);
		flow *= FLOW_STRENGHT;
		newRightMass += flow;
	}

	if (GetType(curSimData, rightBelowIndex) < 2) {
		uint8_t rightBelowMass = GetMass(curSimData, rightBelowIndex);
		uint8_t rightBelowCapacity = clamp((MAX_MASS + (int)rightMass * MAX_COMPRESS) - (int)rightBelowMass, 0, 255);
		int rightBelowFlow = clamp(rightBelowCapacity, -rightBelowMass, rightBelowMass + rightMass <= 255 ? rightMass : 255 - rightBelowMass);
		rightBelowFlow *= FLOW_STRENGHT;
		newRightMass -= rightBelowFlow;
	}

	//end
	//left Update

	if (GetType(curSimData, leftAboveIndex) < 2) {
		uint8_t leftAboveMass = GetMass(curSimData, leftAboveIndex);
		uint8_t leftCurrentCapacity = clamp((MAX_MASS + (int)leftAboveMass * MAX_COMPRESS) - (int)leftMass, 0, 255);
		int flow = clamp(leftCurrentCapacity, -leftMass, leftMass + leftAboveMass <= 255 ? leftAboveMass : 255 - leftMass);
		flow *= FLOW_STRENGHT;
		newLeftMass += flow;
	}

	if (GetType(curSimData, leftBelowIndex) < 2) {
		uint8_t leftBelowMass = GetMass(curSimData, leftBelowIndex);
		uint8_t leftBelowCapacity = clamp((MAX_MASS + (int)leftMass * MAX_COMPRESS) - (int)leftBelowMass, 0, 255);
		int leftBelowFlow = clamp(leftBelowCapacity, -leftBelowMass, leftBelowMass + leftMass <= 255 ? leftMass : 255 - leftBelowMass);
		leftBelowFlow *= FLOW_STRENGHT;
		newLeftMass -= leftBelowFlow;
	}

	//end

	if (belowFlow < 2 || newCurrentMass > MAX_MASS) {
		if (GetType(curSimData, rightIndex) < 2) {
			int rflow = (newRightMass - newCurrentMass) / 2 * FLOW_STRENGHT;
			AddMass(nextSimData, currentIndex, rflow);
		}
		if (GetType(curSimData, leftIndex) < 2) {
			int lflow = (newLeftMass - newCurrentMass) / 2 * FLOW_STRENGHT;
			AddMass(nextSimData, currentIndex, lflow);
		}
	}

	if (GetMass(nextSimData, currentIndex) > MIN_MASS) {
		SetType(nextSimData, currentIndex, 1);
	}
	else {
		SetType(nextSimData, currentIndex, 0);
	}
}

__host__ void* GPU_Alloc(size_t size)
{
	void* ptr = nullptr;
	hipMallocManaged(&ptr, size);
	return ptr;
}

__host__ void GPU_Free(void* ptr) 
{
	hipFree(ptr);
}

__host__ void ComputeSimulation(void* current, void* next, int* size, unsigned int w, unsigned int h) {
	dim3 blockN{ (w * h) / 1024 };
	dim3 threadN{ 1024 };

	ComputeGPU<<<blockN, threadN>>>((uint8_t*)current, (uint8_t*)next, size);

	hipDeviceSynchronize();
}

__host__ void Draw(void* surfaceData, void* simData, unsigned int w, unsigned int h) {
	dim3 blockN{ (w * h) / 1024 };
	dim3 threadN{ 1024 };

	DrawGPU<<<blockN, threadN>>>((uint8_t*)surfaceData, (uint8_t*)simData);

	hipDeviceSynchronize();
}
